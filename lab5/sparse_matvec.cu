#include "hip/hip_runtime.h"
#include <stdio.h>

extern int hipMemcpy();
extern int hipFree();
extern void __syncthreads();
extern int hipMemcpyToSymbol();
void computeGPU(int nr, int *ptr, int *indices, float *b, float *data, float *tgpu);

int compare(float *a, float *b, int size, double threshold)
{
  int i;
  for (i = 0; i < size; i++)
  {
    if (abs(a[i] - b[i]) > threshold)
      return 0;
  }
  return 1;
}

void computeCPU(int nr, int *ptr, int *indices, float *b, float *data, float *tcpu)
{
  int i, j;
  for (i = 0; i < nr; i++)
  {
    for (j = ptr[i]; j < ptr[i + 1]; j++)
    {
      tcpu[i] = tcpu[i] + data[j] * b[indices[j]];
    }
  }
}

void computeGPU(int nr, int *ptr, int *indices, float *b, float *data, float *tgpu)
{
  int i, j;
  for (i = 0; i < nr; i++)
  {
    for (j = ptr[i]; j < ptr[i + 1]; j++)
    {
      tgpu[i] = tgpu[i] + data[j] * b[indices[j]];
    }
  }
}

main(int argc, char **argv)
{
  FILE *fp;
  char line[1024];
  int *ptr, *indices;
  float *data, *b, *tcpu, *tgpu;
  int i;
  int n;  // number of nonzero elements in data
  int nr; // number of rows in matrix
  int nc; // number of columns in matrix

  // Open input file and read to end of comments
  if (argc != 2)
  {
    abort();
  }

  if ((fp = fopen(argv[1], "r")) == NULL)
  {
    abort();
  }

  fgets(line, 128, fp);
  while (line[0] == '%')
  {
    fgets(line, 128, fp);
  }

  // Read number of rows (nr), number of columns (nc) and
  // number of elements and allocate memory for ptr, indices, data, b and t.
  sscanf(line, "%d %d %d\n", &nr, &nc, &n);
  ptr = (int *)malloc((nr + 1) * sizeof(int));
  indices = (int *)malloc(n * sizeof(int));
  data = (float *)malloc(n * sizeof(float));
  b = (float *)malloc(nc * sizeof(float));
  tcpu = (float *)malloc(nr * sizeof(float));
  tgpu = (float *)malloc(nr * sizeof(float));

  // Read data in coordinate format and initialize sparse matrix
  int lastr = 0;
  for (i = 0; i < n; i++)
  {
    int r;
    fscanf(fp, "%d %d %f\n", &r, &(indices[i]), &(data[i]));
    indices[i]--; // start numbering at 0
    if (r != lastr)
    {
      ptr[r - 1] = i;
      lastr = r;
    }
  }
  ptr[nr] = n;

  // initialize t to 0 and b with random data
  for (i = 0; i < nr; i++)
  {
    tcpu[i] = 0.0;
    tgpu[i] = 0.0;
  }
  for (i = 0; i < nc; i++)
  {
    b[i] = (float)rand() / 1111111111;
  }

  // create CUDA event handles for timing purposes
  hipEvent_t start_event, stop_event;
  float elapsed_time_cpu, elapsed_time_gpu;
  elapsed_time_cpu = 0;
  elapsed_time_gpu = 0;

  // Main Computation, CPU version
  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  computeCPU(nr, ptr, indices, b, data, tcpu);
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&elapsed_time_cpu, start_event, stop_event);

  // Main Computation, GPU version
  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  computeGPU(nr, ptr, indices, b, data, tgpu);
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&elapsed_time_gpu, start_event, stop_event);

  // Compare computations to ensure correctness of gpu
  int res = compare(tcpu, tgpu, nr, 0.001);
  if (res == 1)
  {
    printf("VALID!\n  Sequential Time: %.2f msec\n  Parallel Time: %.2f msec\n Speedup = %.2f\n", elapsed_time_cpu, elapsed_time_gpu, elapsed_time_cpu / elapsed_time_gpu);
  }
  else
  {
    printf("INVALID...\n");
  }

  // TODO: Compute result on GPU and compare output
}